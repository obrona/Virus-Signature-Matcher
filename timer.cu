
#include <hip/hip_runtime.h>
#include <string>

struct Timer {
    hipEvent_t begin, end;

    Timer() {
        hipEventCreate(&begin);
        hipEventCreate(&end);
    }

    ~Timer() {
        hipEventDestroy(begin);
        hipEventDestroy(end);
    }


    void start() {
        hipEventRecord(begin);
    }

    void stop() {
        hipEventRecord(end);
    }

    std::string time() {
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, begin, end);
        return std::to_string(milliseconds / 1000);
    }

};