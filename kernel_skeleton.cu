#include "kseq/kseq.h"
#include "common.h"
#include "helpers.cu"
#include "timer.cu"
#include <iostream>

char **samps, **dsamps;
char **phread33, **dphread33;
char **sigs, **dsigs;

double *scores, *dscores; // stores the sum,in int. We get the average later
int *sampLens, *dsampLens;
int *sigLens, *dsigLens;

int MAX;
int ROWS;
int COLS;

void allocMem(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures) {
    samps = new char*[ROWS];
    phread33 = new char*[ROWS];
    sigs = new char*[COLS];
    
    hipMalloc((void**)&dsamps, sizeof(char*) * ROWS);
    hipMalloc((void**)&dphread33, sizeof(char*) * ROWS);
    hipMalloc((void**)&dsigs, sizeof(char*) * COLS);

    scores = new double[MAX];
    sampLens = new int[ROWS];
    sigLens = new int[COLS];

    hipMalloc(&dscores, sizeof(double) * MAX);
    hipMalloc(&dsampLens, sizeof(int) * ROWS);
    hipMalloc(&dsigLens, sizeof(int) * COLS);
    
    
    for (int i = 0; i < ROWS; i ++) {
        hipMalloc(&samps[i], sizeof(char) * samples[i].seq.size());
    }

    for (int i = 0; i < ROWS; i ++) {
        hipMalloc(&phread33[i], sizeof(char) * samples[i].qual.size());
    }

    for (int i = 0; i < COLS; i ++) {
        hipMalloc(&sigs[i], sizeof(char) * signatures[i].seq.size());
    }

   

    for (int i = 0; i < MAX; i ++) scores[i] = -999999.0;

    for (int i = 0; i < ROWS; i ++) sampLens[i] = samples[i].seq.size();

    for (int i = 0; i < COLS; i ++) sigLens[i] = signatures[i].seq.size();

    

    
    
}



void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
    ROWS = samples.size(), COLS = signatures.size();
    MAX = ROWS * COLS;
    
    allocMem(samples, signatures);
    
    for (int i = 0; i < ROWS; i ++) {
        hipMemcpy(samps[i], samples[i].seq.data(), sizeof(char) * samples[i].seq.size(), hipMemcpyHostToDevice);
        hipMemcpy(phread33[i], samples[i].qual.data(), sizeof(char) * samples[i].qual.size(), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < COLS; i ++) {
        hipMemcpy(sigs[i], signatures[i].seq.data(), sizeof(char) * signatures[i].seq.size(), hipMemcpyHostToDevice);
    }
    
    hipMemcpy(dsamps, samps, sizeof(char*) * ROWS, hipMemcpyHostToDevice);
    hipMemcpy(dphread33, phread33, sizeof(char*) * ROWS, hipMemcpyHostToDevice);
    hipMemcpy(dsigs, sigs, sizeof(char*) * COLS, hipMemcpyHostToDevice);
    
    hipMemcpy(dscores, scores, sizeof(double) * MAX, hipMemcpyHostToDevice);
    hipMemcpy(dsampLens, sampLens, sizeof(int) * ROWS, hipMemcpyHostToDevice);
    hipMemcpy(dsigLens, sigLens, sizeof(int) * COLS, hipMemcpyHostToDevice);
    

    dim3 grids = {(unsigned int) ROWS, (unsigned int) COLS, 1};
    combineBoth<<<grids, BLOCK_SIZE>>>(dsamps, dsigs, dphread33, dsampLens, dsigLens, ROWS, COLS, dscores);
    hipMemcpy(scores, dscores, sizeof(double) * MAX, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < ROWS; i ++) {
        for (int j = 0; j < COLS; j ++) {
            int idx = i * COLS + j;
            if (scores[idx] > -1) {
                matches.push_back({samples[i].name, signatures[j].name, scores[idx]});
            }
        }
    }
}
