#include "kseq/kseq.h"
#include "common.h"
#include "helpers.cu"
#include "timer.cu"
#include <iostream>

char **samps, **dsamps;
char **phread33, **dphread33;
char **sigs, **dsigs;

double *scores, *dscores;
int *sampLens, *dsampLens;
int *sigLens, *dsigLens;

int MAX;
int ROWS;
int COLS;

void allocMem(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures) {
    samps = (char**) malloc(sizeof(char*) * ROWS);
    phread33 = (char**) malloc(sizeof(char*) * ROWS);
    sigs = (char**) malloc(sizeof(char*) * COLS);
    
    hipMalloc((void**)&dsamps, sizeof(char*) * ROWS);
    hipMalloc((void**)&dphread33, sizeof(char*) * ROWS);
    hipMalloc((void**)&dsigs, sizeof(char*) * COLS);

    scores = (double*) malloc(sizeof(double) * MAX);
    sampLens = (int*) malloc(sizeof(int) * ROWS);
    sigLens = (int*) malloc(sizeof(int) * COLS);

    hipMalloc(&dscores, sizeof(double) * MAX);
    hipMalloc(&dsampLens, sizeof(int) * ROWS);
    hipMalloc(&dsigLens, sizeof(int) * COLS);
    
    
    for (int i = 0; i < ROWS; i ++) {
        hipMalloc(&samps[i], sizeof(char) * samples[i].seq.size());
    }

    
    for (int i = 0; i < ROWS; i ++) {
        hipMalloc(&phread33[i], sizeof(char) * samples[i].qual.size());
    }

    for (int i = 0; i < COLS; i ++) {
        hipMalloc(&sigs[i], sizeof(char) * signatures[i].seq.size());
    }

   

    for (int i = 0; i < MAX; i ++) scores[i] = -999999.0;

    for (int i = 0; i < ROWS; i ++) sampLens[i] = samples[i].seq.size();

    for (int i = 0; i < COLS; i ++) sigLens[i] = signatures[i].seq.size();

    

    
    
}



void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
    ROWS = samples.size(), COLS = signatures.size();
    MAX = ROWS * COLS;
    
    allocMem(samples, signatures);
    
    for (int i = 0; i < ROWS; i ++) {
        hipMemcpy(samps[i], samples[i].seq.data(), sizeof(char) * samples[i].seq.size(), hipMemcpyHostToDevice);
        hipMemcpy(phread33[i], samples[i].qual.data(), sizeof(char) * samples[i].qual.size(), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < COLS; i ++) {
        hipMemcpy(sigs[i], signatures[i].seq.data(), sizeof(char) * signatures[i].seq.size(), hipMemcpyHostToDevice);
    }
    
    hipMemcpy(dsamps, samps, sizeof(char*) * ROWS, hipMemcpyHostToDevice);
    hipMemcpy(dphread33, phread33, sizeof(char*) * ROWS, hipMemcpyHostToDevice);
    hipMemcpy(dsigs, sigs, sizeof(char*) * COLS, hipMemcpyHostToDevice);
    
    hipMemcpy(dscores, scores, sizeof(double) * MAX, hipMemcpyHostToDevice);
    hipMemcpy(dsampLens, sampLens, sizeof(int) * ROWS, hipMemcpyHostToDevice);
    hipMemcpy(dsigLens, sigLens, sizeof(int) * COLS, hipMemcpyHostToDevice);
    

    int NUM_BLKS = (MAX + BLOCK_SIZE) / BLOCK_SIZE;
    matcherKernel<<<NUM_BLKS, BLOCK_SIZE>>>(dsamps, dsigs, dphread33, dsampLens, dsigLens, ROWS, COLS, dscores);
    hipMemcpy(scores, dscores, sizeof(double) * MAX, hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    
    for (int i = 0; i < ROWS; i ++) {
        for (int j = 0; j < COLS; j ++) {
            int idx = i * COLS + j;
            if (scores[idx] > -0.000001) {
                matches.push_back({samples[i].name, signatures[j].name, scores[idx]});
            }
        }
    }
}
